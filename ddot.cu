#include "hip/hip_runtime.h"
#include "ddot.hpp"
#include <stdio.h>

__global__ void dot(double * x, double * y, double * z, int n) {
	int index = threadIdx.x + blockIdx.x *blockDim.x;
	if (index < n) {
		z[index] = x[index] * y[index];
	}
	
	__syncthreads();
}

int ddot (const int n, const double * const x, const double * const y, 
	  double * const result, double & time_allreduce)
{
	double * z = (double*)malloc(n * sizeof(double));
	int threadsPerBlock = 512;
	int blocksPerGrid = (n + threadsPerBlock - 1)/threadsPerBlock;
	double * x_d = NULL;
	hipMalloc((void**)&x_d, n*sizeof(double));
	double * y_d = NULL;
	hipMalloc((void**)&y_d, n*sizeof(double));
	double * z_d = NULL;
	hipMalloc((void**)&z_d, n*sizeof(double));
	hipMemcpy(x_d, x, n*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(y_d, y, n*sizeof(double), hipMemcpyHostToDevice);
	dot<<<blocksPerGrid, 512>>>(x_d, y_d, z_d, n);
	hipDeviceSynchronize();
	hipMemcpy(z, z_d, n*sizeof(double), hipMemcpyDeviceToHost);
	hipFree(x_d);
	hipFree(y_d);
	hipFree(z_d);
	double sum = 0;
	for (int i = 0; i < n; i++) {
		sum += z[i];
	}
	free(z);
	*result = sum;
	//printf("What is %f \n", *result);
	hipDeviceReset();	
	return(0);
}
